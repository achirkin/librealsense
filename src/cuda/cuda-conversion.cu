
#include <hip/hip_runtime.h>
#ifdef RS2_USE_CUDA

#include "cuda-conversion.cuh"
#include <iostream>
#include <iomanip>

__global__ void kernel_unpack_yuy2_y8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= superPixCount)
		return;

	int idx = i * 4;
	
	dst[idx] = src[idx];
	dst[idx + 1] = src[idx + 2];
	dst[idx + 2] = src[idx + 4];
	dst[idx + 3] = src[idx + 6];
	dst[idx + 4] = src[idx + 8];
	dst[idx + 5] = src[idx + 10];
	dst[idx + 6] = src[idx + 12];
    dst[idx + 7] = src[idx + 14];
    dst[idx + 8] = src[idx + 16];
    dst[idx + 9] = src[idx + 18];
    dst[idx + 10] = src[idx + 20];
    dst[idx + 11] = src[idx + 22];
    dst[idx + 12] = src[idx + 24];
    dst[idx + 13] = src[idx + 26];
    dst[idx + 14] = src[idx + 28];
    dst[idx + 15] = src[idx + 30];
}

__global__ void kernel_unpack_yuy2_y16_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= superPixCount)
		return;


	int idx = i * 4;

	dst[idx] = 0;
	dst[idx + 1] = src[idx + 0];
	dst[idx + 2] = 0;
	dst[idx + 3] = src[idx + 2];
	dst[idx + 4] = 0;
	dst[idx + 5] = src[idx + 4];
	dst[idx + 6] = 0;
	dst[idx + 7] = src[idx + 6];
	dst[idx + 8] = 0;
	dst[idx + 9] = src[idx + 8];
	dst[idx + 10] = 0;
	dst[idx + 11] = src[idx + 10];
	dst[idx + 12] = 0;
	dst[idx + 13] = src[idx + 12];
	dst[idx + 14] = 0;
    dst[idx + 15] = src[idx + 14];
    dst[idx + 16] = 0;
    dst[idx + 17] = src[idx + 16];
    dst[idx + 18] = 0;
    dst[idx + 19] = src[idx + 18];
    dst[idx + 20] = 0;
    dst[idx + 21]= src[idx + 20];
    dst[idx + 22] = 0;
    dst[idx + 23] = src[idx + 22];
    dst[idx + 24] = 0;
    dst[idx + 25] = src[idx + 24];
    dst[idx + 26] = 0;
    dst[idx + 27] = src[idx + 26];
    dst[idx + 28] = 0;
    dst[idx + 29] = src[idx + 28];
    dst[idx + 30] = 0;
    dst[idx + 31] = src[idx + 30];
}

__global__ void kernel_unpack_yuy2_rgb8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
 //   int stride = blockDim.x * gridDim.x;
    
	if (i >= superPixCount)
		return;
		
//	for (int j = i; j < superPixCount; j += stride) {

	    int idx = i * 4;

	    uint8_t y0 = src[idx];
	    uint8_t u0 = src[idx + 1];
	    uint8_t y1 = src[idx + 2];
	    uint8_t v0 = src[idx + 3];

	    int16_t c = y0 - 16;
	    int16_t d = u0 - 128;
	    int16_t e = v0 - 128;

	    int32_t t;
    #define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

	    int odx = i * 6;

	    dst[odx] = clamp((298 * c + 409 * e + 128) >> 8);
	    dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	    dst[odx + 2] = clamp((298 * c + 516 * d + 128) >> 8);

	    c = y1 - 16;

	    dst[odx + 3] = clamp((298 * c + 409 * e + 128) >> 8);
	    dst[odx + 4] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	    dst[odx + 5] = clamp((298 * c + 516 * d + 128) >> 8);

    #undef clamp

 //   }
}

__global__ void kernel_unpack_yuy2_bgr8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= superPixCount)
		return;

	int idx = i * 4;

	uint8_t y0 = src[idx];
	uint8_t u0 = src[idx + 1];
	uint8_t y1 = src[idx + 2];
	uint8_t v0 = src[idx + 3];

	int16_t c = y0 - 16;
	int16_t d = u0 - 128;
	int16_t e = v0 - 128;

	int32_t t;
#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

	int odx = i * 6;

	dst[odx + 2] = clamp((298 * c + 409 * e + 128) >> 8);
	dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	dst[odx    ] = clamp((298 * c + 516 * d + 128) >> 8);

	c = y1 - 16;

	dst[odx + 5] = clamp((298 * c + 409 * e + 128) >> 8);
	dst[odx + 4] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	dst[odx + 3] = clamp((298 * c + 516 * d + 128) >> 8);

#undef clamp
}


__global__ void kernel_unpack_yuy2_rgba8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= superPixCount)
		return;

	int idx = i * 4;

	uint8_t y0 = src[idx];
	uint8_t u0 = src[idx + 1];
	uint8_t y1 = src[idx + 2];
	uint8_t v0 = src[idx + 3];

	int16_t c = y0 - 16;
	int16_t d = u0 - 128;
	int16_t e = v0 - 128;

	int32_t t;
#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

	int odx = i * 8;

	dst[odx] = clamp((298 * c + 409 * e + 128) >> 8);
	dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	dst[odx + 2] = clamp((298 * c + 516 * d + 128) >> 8);
	dst[odx + 3] = 255;

	c = y1 - 16;

	dst[odx + 4] = clamp((298 * c + 409 * e + 128) >> 8);
	dst[odx + 5] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	dst[odx + 6] = clamp((298 * c + 516 * d + 128) >> 8);
	dst[odx + 7] = 255;

#undef clamp
}

__global__ void kernel_unpack_yuy2_bgra8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= superPixCount)
		return;

	int idx = i * 4;

	uint8_t y0 = src[idx];
	uint8_t u0 = src[idx + 1];
	uint8_t y1 = src[idx + 2];
	uint8_t v0 = src[idx + 3];

	int16_t c = y0 - 16;
	int16_t d = u0 - 128;
	int16_t e = v0 - 128;

	int32_t t;
#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

	int odx = i * 8;

    dst[odx + 3] = 255;
	dst[odx + 2] = clamp((298 * c + 409 * e + 128) >> 8);
	dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	dst[odx    ] = clamp((298 * c + 516 * d + 128) >> 8);

	c = y1 - 16;

    dst[odx + 7] = 255;
	dst[odx + 6] = clamp((298 * c + 409 * e + 128) >> 8);
	dst[odx + 5] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	dst[odx + 4] = clamp((298 * c + 516 * d + 128) >> 8);

#undef clamp
}


void rsimpl::unpack_yuy2_cuda_helper(const uint8_t* src, uint8_t* dst, int n, rs2_format format) 
{

   // cudaEvent_t start_core, stop_core;
	// cudaEventCreate(&start_core);
    // cudaEventCreate(&stop_core);
    
   // cudaEvent_t start, stop;
	//cudaEventCreate(&start);
  //  cudaEventCreate(&stop);
    
   // cudaEvent_t start_mem, stop_mem;
   // cudaEventCreate(&start_mem);
   // cudaEventCreate(&stop_mem);
   // float mem_ms_in, mem_ms_out;
	
	//cudaEventRecord(start);
    
	// How many super pixels do we have?
	int superPix = n / 2;
	uint8_t *devSrc = 0;
	uint8_t *devDst = 0;
	
	//uint8_t *host_dst;
	//cudaError_t result = cudaMallocHost(&host_dst, n * sizeof(uint8_t) * 3);

	cudaError_t result = cudaMalloc(&devSrc, superPix * sizeof(uint8_t) * 4);
	assert(result == cudaSuccess);
	
	result = cudaMemcpy(devSrc, src, superPix * sizeof(uint8_t) * 4, cudaMemcpyHostToDevice);
	assert(result == cudaSuccess);
	
	int numBlocks = superPix / RS2_CUDA_THREADS_PER_BLOCK;
	int size;

	switch (format)
	{
	case RS2_FORMAT_Y8: // 1
		size = 1;
		result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_y8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK >>>(devSrc, devDst, superPix);
		break;
	case RS2_FORMAT_Y16: // 2
		size = 2;
		result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_y16_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK >>>(devSrc, devDst, superPix);
		break;
	case RS2_FORMAT_RGB8:
	    size = 3;
		result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_rgb8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK >>>(devSrc, devDst, superPix);
		break;
	case RS2_FORMAT_BGR8:
	    size = 3;
        result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_bgr8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK >>>(devSrc, devDst, superPix);
    	break;
	case RS2_FORMAT_RGBA8: // 4
		size = 4;
		result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_rgba8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK >>>(devSrc, devDst, superPix);
		break;
	case RS2_FORMAT_BGRA8: // 4
        size = 4;
		result = cudaMalloc(&devDst, n * sizeof(uint8_t) * size);
	    assert(result == cudaSuccess);
		kernel_unpack_yuy2_bgra8_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK >>>(devSrc, devDst, superPix);
		break;
	default:
		assert(false);
	}

	result = cudaGetLastError();
	assert(result == cudaSuccess);
    result = cudaMemcpy(dst, devDst, n * sizeof(uint8_t) * size, cudaMemcpyDeviceToHost);
	assert(result == cudaSuccess);


    // copy test
    /*
    cudaEventRecord(start_mem);
	result = cudaMemcpy(host_dst, devDst, n * sizeof(uint8_t) * 3, cudaMemcpyDeviceToHost);
	assert(result == cudaSuccess);
	cudaEventRecord(stop_mem);
	cudaEventSynchronize(stop_mem);
	cudaEventElapsedTime(&mem_ms_tst_out, start_mem, stop_mem);
    */

	// Copy back
//	cudaEventRecord(start_mem);

//    memcpy(dst, host_dst, n * sizeof(uint8_t) * 3);
/*	cudaEventRecord(stop_mem);
	cudaEventSynchronize(stop_mem);
	cudaEventElapsedTime(&mem_ms_out, start_mem, stop_mem);
*/
	cudaFree(devSrc);
	cudaFree(devDst);
	
	//cudaFreeHost(host_dst);
	
	//cudaEventRecord(stop);
	
//	cudaEventSynchronize(stop);
   // float milliseconds = 0;
  //  cudaEventElapsedTime(&ms_core, start_core, stop_core);
  //  std::cout << std::setprecision(5);
//    std::cout << "kernel: " << ms_core;
   // cudaEventElapsedTime(&milliseconds, start, stop);
//    std::cout << " all:" << milliseconds;
//    std::cout << " copies (in, out a, out b): " << mem_ms_in << ", " << mem_ms_tst_out << ", " << mem_ms_out << std::endl;
//    std::cout << " ratio: " << mem_ms_in/(mem_ms_in+mem_ms_out) << std::endl;
}


__global__ void kernel_split_frame_y8_y8_from_y8i_cuda(uint8_t* a, uint8_t* b, int count, const rsimpl::y8i_pixel * source)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= count)
		return;

    a[i] = source[i].l;
    b[i] = source[i].r;
}

void rsimpl::y8_y8_from_y8i_cuda_helper(uint8_t* const dest[], int count, const rsimpl::y8i_pixel * source)
{
    
//    cudaEvent_t start, stop;
//	cudaEventCreate(&start);
//    cudaEventCreate(&stop);
    	
//	cudaEventRecord(start);    
    
    cudaStream_t stream;
    cudaStreamCreate(&stream);

    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    uint8_t* a = dest[0];
    uint8_t* b = dest[1];
    
    rsimpl::y8i_pixel *devSrc = 0;
    uint8_t *devDst1 = 0; // for dest[0]
    uint8_t *devDst2 = 0; // for dest[1]
    
    cudaError_t result = cudaMalloc(&devSrc, count * sizeof(rsimpl::y8i_pixel));
    assert(result == cudaSuccess);

    result = cudaMemcpyAsync(devSrc, source, count * sizeof(rsimpl::y8i_pixel), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);

    result = cudaMalloc(&devDst1, count * sizeof(uint8_t));
    assert(result == cudaSuccess);

    result = cudaMalloc(&devDst2, count * sizeof(uint8_t));
    assert(result == cudaSuccess);
    
    //no generic function
    kernel_split_frame_y8_y8_from_y8i_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK,0,stream>>>(devDst1, devDst2, count, devSrc);

    result = cudaGetLastError();
    assert(result == cudaSuccess);
    
    result = cudaMemcpyAsync(a, devDst1, count * sizeof(uint8_t), cudaMemcpyDeviceToHost, stream);
    assert(result == cudaSuccess);
    result = cudaMemcpyAsync(b, devDst2, count * sizeof(uint8_t), cudaMemcpyDeviceToHost, stream);
    assert(result == cudaSuccess);
    cudaStreamSynchronize(stream);
    cudaStreamDestroy(stream);

    cudaFree(devSrc);
    cudaFree(devDst1);
    cudaFree(devDst2);
    
    	
//	cudaEventRecord(stop);
	
//	cudaEventSynchronize(stop);
//    float milliseconds = 0;
  //  std::cout << std::setprecision(5);
//    cudaEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "time: " << milliseconds << std::endl;
}

__global__ void kernel_split_frame_y16_y16_from_y12i_cuda(uint16_t* a, uint16_t* b, int count, const rsimpl::y12i_pixel * source)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= count)
		return;

    a[i] = source[i].l() << 6 | source[i].l() >> 4;
    b[i] = source[i].r() << 6 | source[i].r() >> 4;
}


void rsimpl::y16_y16_from_y12i_10_cuda_helper(uint8_t* const dest[], int count, const rsimpl::y12i_pixel * source)
{
    
//   source =  reinterpret_cast<const y12i_pixel*>(source);
 
//    cudaEvent_t start, stop;
//	cudaEventCreate(&start);
//    cudaEventCreate(&stop);
    	
//	cudaEventRecord(start);    
    
    cudaStream_t stream;
    cudaStreamCreate(&stream);

    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    uint16_t* a = reinterpret_cast<uint16_t*>(dest[0]);
    uint16_t* b = reinterpret_cast<uint16_t*>(dest[1]);
    
    rsimpl::y12i_pixel *devSrc = 0;
    uint16_t *devDst1 = 0; // for dest[0]
    uint16_t *devDst2 = 0; // for dest[1]
    
    cudaError_t result = cudaMalloc(&devSrc, count * sizeof(rsimpl::y12i_pixel));
    assert(result == cudaSuccess);

    result = cudaMemcpyAsync(devSrc, source, count * sizeof(rsimpl::y12i_pixel), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);

    result = cudaMalloc(&devDst1, count * sizeof(uint16_t));
    assert(result == cudaSuccess);

    result = cudaMalloc(&devDst2, count * sizeof(uint16_t));
    assert(result == cudaSuccess);
    
    //no generic function
    kernel_split_frame_y16_y16_from_y12i_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK,0,stream>>>(devDst1, devDst2, count, devSrc);

    result = cudaGetLastError();
    assert(result == cudaSuccess);
    
    result = cudaMemcpyAsync(a, devDst1, count * sizeof(uint16_t), cudaMemcpyDeviceToHost, stream);
    assert(result == cudaSuccess);
    result = cudaMemcpyAsync(b, devDst2, count * sizeof(uint16_t), cudaMemcpyDeviceToHost, stream);
    assert(result == cudaSuccess);
    cudaStreamSynchronize(stream);
    cudaStreamDestroy(stream);

    cudaFree(devSrc);
    cudaFree(devDst1);
    cudaFree(devDst2);
    
    	
//	cudaEventRecord(stop);
	
//	cudaEventSynchronize(stop);
//    float milliseconds = 0;
  //  std::cout << std::setprecision(5);
//    cudaEventElapsedTime(&milliseconds, start, stop);
//    std::cout << "time: " << milliseconds << std::endl;
}


__global__ void kernel_z16_y8_from_sr300_inzi_cuda (const uint16_t* source, uint8_t* const dest, int count)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= count)
		return;

    dest[i] = source[i] >> 2;
}

void rsimpl::unpack_z16_y8_from_sr300_inzi_cuda (uint8_t * const dest, const uint16_t * source, int count) 
{
    uint16_t *devSrc = 0;
    uint8_t *devDst = 0;
     
    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    
    cudaError_t result = cudaMalloc(&devSrc, count * sizeof(uint16_t));
    assert(result == cudaSuccess);

    result = cudaMemcpy(devSrc, source, count * sizeof(uint16_t), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);
    
    result = cudaMalloc(&devDst, count * sizeof(uint8_t));
    assert(result == cudaSuccess);

    kernel_z16_y8_from_sr300_inzi_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, count);
    
    result = cudaMemcpy(dest, devDst, count * sizeof(uint8_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);
    
    cudaFree(devSrc);
    cudaFree(devDst);

}

__global__ void kernel_z16_y16_from_sr300_inzi_cuda (uint16_t* const source, uint16_t* const dest, int count)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= count)
		return;

    dest[i] = source[i] << 6;
}

void rsimpl::unpack_z16_y16_from_sr300_inzi_cuda(uint16_t * const dest, const uint16_t * source, int count) 
{
    uint16_t *devSrc = 0;
    uint16_t *devDst = 0;
     
    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    
    cudaError_t result = cudaMalloc(&devSrc, count * sizeof(uint16_t));
    assert(result == cudaSuccess);

    result = cudaMemcpy(devSrc, source, count * sizeof(uint16_t), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);
    
    result = cudaMalloc(&devDst, count * sizeof(uint16_t));
    assert(result == cudaSuccess);

    kernel_z16_y16_from_sr300_inzi_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devSrc, devDst, count);
    
    result = cudaMemcpy(dest, devDst, count * sizeof(uint16_t), cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);
    
    cudaFree(devSrc);
    cudaFree(devDst);
}

#endif
