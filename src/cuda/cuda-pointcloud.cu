
#include <hip/hip_runtime.h>
#ifdef RS2_USE_CUDA

#include "cuda-pointcloud.cuh"
#include <iostream>


__device__
float map_depth (float depth_scale, uint16_t z) {
    return depth_scale * z;
}

__device__
void deproject_pixel_to_point_cuda(float points[3], const struct rs2_intrinsics * intrin, const float pixel[2], float depth) {
    assert(intrin->model != RS2_DISTORTION_MODIFIED_BROWN_CONRADY); // Cannot deproject from a forward-distorted image
    assert(intrin->model != RS2_DISTORTION_FTHETA); // Cannot deproject to an ftheta image
    //assert(intrin->model != RS2_DISTORTION_BROWN_CONRADY); // Cannot deproject to an brown conrady model
    float x = (pixel[0] - intrin->ppx) * intrin->fx;
    float y = (pixel[1] - intrin->ppy) * intrin->fy;
    if(intrin->model == RS2_DISTORTION_INVERSE_BROWN_CONRADY)
    {
        float r2  = x*x + y*y;
        float f = 1 + intrin->coeffs[0]*r2 + intrin->coeffs[1]*r2*r2 + intrin->coeffs[4]*r2*r2*r2;
        float ux = x*f + 2*intrin->coeffs[2]*x*y + intrin->coeffs[3]*(r2 + 2*x*x);
        float uy = y*f + 2*intrin->coeffs[3]*x*y + intrin->coeffs[2]*(r2 + 2*y*y);
        x = ux;
        y = uy;
    }
    points[0] = depth * x;
    points[1] = depth * y;
    points[2] = depth;
    
}


__global__
//void kernel_deproject_depth_cuda(float * points, const rs2_intrinsics & intrin, const uint16_t * depth, std::function<uint16_t(float)> map_depth)

void kernel_deproject_depth_cuda(float * points, const rs2_intrinsics* intrin, const uint16_t * depth, float depth_scale)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= (*intrin).height * (*intrin).width)
        return;
    int stride = blockDim.x * gridDim.x;
    int a, b;

    for (int j = i; j < (*intrin).height * (*intrin).width; j += stride) {
        
    //    x = (blockIdx.x * blockDim.x) + threadIdx.x;
     //   y = (blockIdx.y * blockDim.y) + threadIdx.y;
        
     //   printf("x, y: %d, %d\n", x, y);
        
    //    if (x >= (*intrin).height || y >= (*intrin).width) return;
        b = floorf( j / (*intrin).width );
        a = j - b * (*intrin).width;
        const float pixel[] = { (float)a, (float)b };
        deproject_pixel_to_point_cuda(points + j * 3, intrin, pixel, map_depth(depth_scale, depth[j]));                     
   }
}



void rsimpl::deproject_depth_cuda(float * points, const rs2_intrinsics & intrin, const uint16_t * depth, float depth_scale)
{
    int count = intrin.height * intrin.width;
    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    
 //   std::cout << "image size: " << intrin.height << "*" << intrin.width << std::endl;
    
    float *dev_points = 0;	
    uint16_t *dev_depth = 0;
    rs2_intrinsics* dev_intrin = 0;
    
  //   float *pinned_points = 0;	
    
 //   cudaStream_t streams[3];
    cudaError_t result;
    cudaStream_t stream1;
    cudaStream_t stream2;
 //   size_t pitch;
    
    cudaStreamCreate(&stream1);
    cudaStreamCreate(&stream2);
   
  //  result = cudaMallocHost(&pinned_points, count * sizeof(float) * 3);
  //  assert(result == cudaSuccess);
    
    result = cudaMalloc(&dev_points, count * sizeof(float) * 3);
 //   result = cudaMallocPitch(&dev_points, &pitch, intrin.width * sizeof(float) * 3, intrin.height);
    assert(result == cudaSuccess);
    
    result = cudaMalloc(&dev_depth, count * sizeof(uint16_t));
    assert(result == cudaSuccess);
   
    
    result = cudaMemcpyAsync(dev_depth, depth, count * sizeof(uint16_t), cudaMemcpyHostToDevice, stream1);
    result = cudaMemcpy(dev_depth, depth, count * sizeof(uint16_t), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);

    result = cudaMalloc(&dev_intrin, sizeof(rs2_intrinsics));
    assert(result == cudaSuccess);

    result = cudaMemcpyAsync(dev_intrin, &intrin, sizeof(rs2_intrinsics), cudaMemcpyHostToDevice, stream2);
    result = cudaMemcpy(dev_intrin, &intrin, sizeof(rs2_intrinsics), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);
    
    
    dev_intrin->fx = 1 / dev_intrin->fx;
    dev_intrin->fy = 1 / dev_intrin->fy;
    
  
    cudaStreamSynchronize(stream2);
    cudaStreamSynchronize(stream1);

    
    kernel_deproject_depth_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(dev_points, dev_intrin, dev_depth, depth_scale); 

    result = cudaMemcpy(points, dev_points, count * sizeof(float) * 3, cudaMemcpyDeviceToHost);
    memcpy(points, points, count * sizeof(float) * 3);
  
//  result = cudaMemcpy(pinned_points, dev_points, count * sizeof(float) * 3, cudaMemcpyDeviceToHost);
    
  //  cudaMemcpy2D(points, intrin.width * sizeof(float) * 3, dev_points, pitch, intrin.width * sizeof(float) * 3, intrin.height, cudaMemcpyDeviceToHost);
  //  printf("result: %d \n", result);
    assert(result == cudaSuccess);
 /*   
    for (int i = 0; i < 3; i++) 
    {
        result = cudaStreamDestroy(streams[i]);
    }    
    */
    
    result = cudaStreamDestroy(stream1);
    result = cudaStreamDestroy(stream2);
    assert(result == cudaSuccess);
    
    cudaFree(dev_points);
    cudaFree(dev_depth);
    cudaFree(dev_intrin);
}

#endif
