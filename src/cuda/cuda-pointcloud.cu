
#include <hip/hip_runtime.h>
#ifdef RS2_USE_CUDA

#include "cuda-pointcloud.cuh"



__host__ __device__
void deproject_pixel_to_point_cuda(float point[3], const struct rs2_intrinsics * intrin, const float pixel[2], float depth) {
    assert(intrin->model != RS2_DISTORTION_MODIFIED_BROWN_CONRADY); // Cannot deproject from a forward-distorted image
    assert(intrin->model != RS2_DISTORTION_FTHETA); // Cannot deproject to an ftheta image
    //assert(intrin->model != RS2_DISTORTION_BROWN_CONRADY); // Cannot deproject to an brown conrady model

    float x = (pixel[0] - intrin->ppx) / intrin->fx;
    float y = (pixel[1] - intrin->ppy) / intrin->fy;
    if(intrin->model == RS2_DISTORTION_INVERSE_BROWN_CONRADY)
    {
        float r2  = x*x + y*y;
        float f = 1 + intrin->coeffs[0]*r2 + intrin->coeffs[1]*r2*r2 + intrin->coeffs[4]*r2*r2*r2;
        float ux = x*f + 2*intrin->coeffs[2]*x*y + intrin->coeffs[3]*(r2 + 2*x*x);
        float uy = y*f + 2*intrin->coeffs[3]*x*y + intrin->coeffs[2]*(r2 + 2*y*y);
        x = ux;
        y = uy;
    }
    point[0] = depth * x;
    point[1] = depth * y;
    point[2] = depth;
}

template<class MAP_DEPTH>
__global__
void kernel_deproject_depth_cuda(float * points, const rs2_intrinsics & intrin, const uint16_t * depth, std::function<uint16_t(float)> map_depth)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    deproject_pixel_to_point_cuda(points[i * 3], &intrin, pixel, map_depth(depth[i]); 
    /*
    
    for (int y = 0; y<intrin.height; ++y)
    {
        for (int x = 0; x<intrin.width; ++x)
        {
            const float pixel[] = { (float)x, (float)y };
            rs2_deproject_pixel_to_point(points, &intrin, pixel, map_depth(*depth++));
            points += 3;
        }
    }
    */
}



void deproject_depth_cuda(float * points, const rs2_intrinsics & intrin, const uint16_t * depth, std::function<uint16_t(float)> map_depth)
{
// need to copy: points, intrin?, depth
    int count = intrin.height * intrin.width;
    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    
    float *devPoints = 0;	
	uint16_t *devDepth = 0;

	cudaError_t result = cudaMalloc(&devPoints, count * sizeof(float) * 3);
	assert(result == cudaSuccess);
	
	result = cudaMemcpy(devPoints, points, count * sizeof(float) * 3, cudaMemcpyHostToDevice);
	assert(result == cudaSuccess);
	
    result = cudaMalloc(&devDepth, count * sizeof(float));
	assert(result == cudaSuccess);
	
	result = cudaMemcpy(devDepth, depth, count * sizeof(float), cudaMemcpyHostToDevice);
	assert(result == cudaSuccess);
	
	kernel_deproject_depth_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(devPoints, intrin, depth, map_depth);
    
    result = cudaMemcpy(points, devPoints, count * sizeof(float) * 3, cudaMemcpyHostToDevice);
	assert(result == cudaSuccess);
        
    cudaFree(devPoints);
    cudaFree(devDepth);
}

#endif
