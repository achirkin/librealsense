
#include <hip/hip_runtime.h>
#ifdef RS2_USE_CUDA

#include "cuda-pointcloud.cuh"
#include <iostream>
#include <chrono>


__device__
float map_depth (float depth_scale, uint16_t z) {
    return depth_scale * z;
}

__device__
void deproject_pixel_to_point_cuda(float points[3], const struct rs2_intrinsics * intrin, const float pixel[2], float depth) {
    assert(intrin->model != RS2_DISTORTION_MODIFIED_BROWN_CONRADY); // Cannot deproject from a forward-distorted image
    assert(intrin->model != RS2_DISTORTION_FTHETA); // Cannot deproject to an ftheta image
    //assert(intrin->model != RS2_DISTORTION_BROWN_CONRADY); // Cannot deproject to an brown conrady model
    float x = (pixel[0] - intrin->ppx) * intrin->fx;
    float y = (pixel[1] - intrin->ppy) * intrin->fy;    
    if(intrin->model == RS2_DISTORTION_INVERSE_BROWN_CONRADY)
    {
        float r2  = x*x + y*y;
        float f = 1 + intrin->coeffs[0]*r2 + intrin->coeffs[1]*r2*r2 + intrin->coeffs[4]*r2*r2*r2;
        float ux = x*f + 2*intrin->coeffs[2]*x*y + intrin->coeffs[3]*(r2 + 2*x*x);
        float uy = y*f + 2*intrin->coeffs[3]*x*y + intrin->coeffs[2]*(r2 + 2*y*y);
        x = ux;
        y = uy;
    } 
    points[0] = depth * x;
    points[1] = depth * y;
    points[2] = depth;
    
}


__global__
//void kernel_deproject_depth_cuda(float * points, const rs2_intrinsics & intrin, const uint16_t * depth, std::function<uint16_t(float)> map_depth)

void kernel_deproject_depth_cuda(float * points, const rs2_intrinsics* intrin, const uint16_t * depth, float depth_scale)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i >= (*intrin).height * (*intrin).width) {
        printf("bye");
        return;
    }
    int stride = blockDim.x * gridDim.x;
    int a, b;
    
    for (int j = i; j < (*intrin).height * (*intrin).width; j += stride) {
        b = j / (*intrin).width;
        a = j - b * (*intrin).width;
        const float pixel[] = { (float)a, (float)b };
        deproject_pixel_to_point_cuda(points + j * 3, intrin, pixel, depth_scale * depth[j]);               
   }
}


void rsimpl::deproject_depth_cuda(float * points, const rs2_intrinsics & intrin, const uint16_t * depth, float depth_scale)
{
    int count = intrin.height * intrin.width;
    int numBlocks = count / RS2_CUDA_THREADS_PER_BLOCK;
    
    float *dev_points = 0;	
    uint16_t *dev_depth = 0;
  //  float* temp_x;
  //  float* temp_y;
    rs2_intrinsics* dev_intrin = 0;
    
    cudaEvent_t start1, start2, start3, start4, stop1, stop2, stop3, stop4;
    float elapsedTime;
    
  //   float *pinned_points = 0;	
    
 //   cudaStream_t streams[3];
    cudaError_t result;
  //  cudaStream_t stream1;
  //  cudaStream_t stream2;
 //   size_t pitch;
    
  //  cudaStreamCreate(&stream1);
  //  cudaStreamCreate(&stream2);
   
  //  result = cudaMallocHost(&pinned_points, count * sizeof(float) * 3);
  //  assert(result == cudaSuccess);
      /* 
     cudaEventCreate(&start1);
     cudaEventRecord(start1,0);
    */
    result = cudaMalloc(&dev_points, count * sizeof(float) * 3);
 //   result = cudaMallocPitch(&dev_points, &pitch, intrin.width * sizeof(float) * 3, intrin.height);
    assert(result == cudaSuccess);
  /*  
     cudaEventCreate(&stop1);
     cudaEventRecord(stop1,0);
     cudaEventSynchronize(stop1);

     cudaEventElapsedTime(&elapsedTime, start1,stop1);
     printf("malloc : %f ms\n" ,elapsedTime);
*/
    
    result = cudaMalloc(&dev_depth, count * sizeof(uint16_t));
    assert(result == cudaSuccess);
    
   // result = cudaMalloc(&temp_x, count * sizeof(float));
   // assert(result == cudaSuccess);
    
   // result = cudaMalloc(&temp_y, count * sizeof(float));
   // assert(result == cudaSuccess);
    
   /*    
     cudaEventCreate(&start2);
     cudaEventRecord(start2,0);
    */
 //   result = cudaMemcpyAsync(dev_depth, depth, count * sizeof(uint16_t), cudaMemcpyHostToDevice, stream1);
    result = cudaMemcpy(dev_depth, depth, count * sizeof(uint16_t), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess);
 /*   
     cudaEventCreate(&stop2);
     cudaEventRecord(stop2,0);
     cudaEventSynchronize(stop2);

     cudaEventElapsedTime(&elapsedTime, start2,stop2);
     printf("host to device : %f ms\n" ,elapsedTime);
*/
    result = cudaMalloc(&dev_intrin, sizeof(rs2_intrinsics));
    assert(result == cudaSuccess);


    rs2_intrinsics intrin_cpy = rs2_intrinsics(intrin);
    intrin_cpy.fx = 1 / intrin_cpy.fx;
    intrin_cpy.fy = 1 / intrin_cpy.fy;


  //  result = cudaMemcpyAsync(dev_intrin, &intrin, sizeof(rs2_intrinsics), cudaMemcpyHostToDevice, stream2);
    result = cudaMemcpy(dev_intrin, &intrin_cpy, sizeof(rs2_intrinsics), cudaMemcpyHostToDevice);
    assert(result == cudaSuccess); 
  
 //   cudaStreamSynchronize(stream2);
//    cudaStreamSynchronize(stream1);
    /*
     cudaEventCreate(&start3);
     cudaEventRecord(start3,0);
    */
     kernel_deproject_depth_cuda<<<numBlocks, RS2_CUDA_THREADS_PER_BLOCK>>>(dev_points, dev_intrin, dev_depth, depth_scale); 
    /*
     cudaEventCreate(&stop3);
     cudaEventRecord(stop3,0);
     cudaEventSynchronize(stop3);

     cudaEventElapsedTime(&elapsedTime, start3,stop3);
     printf("kernel : %f ms\n" ,elapsedTime);
    */   
/*
     cudaEventCreate(&start4);
     cudaEventRecord(start4,0);
   */ 
    result = cudaMemcpy(points, dev_points, count * sizeof(float) * 3, cudaMemcpyDeviceToHost);
    assert(result == cudaSuccess);
/*     
     cudaEventCreate(&stop4);
     cudaEventRecord(stop4,0);
     cudaEventSynchronize(stop4);
     
     cudaEventElapsedTime(&elapsedTime, start4,stop4);
     printf("device to host : %f ms\n" ,elapsedTime);
     */   

    

//    memcpy(points, points, count * sizeof(float) * 3);
  
//  result = cudaMemcpy(pinned_points, dev_points, count * sizeof(float) * 3, cudaMemcpyDeviceToHost);
    
  //  cudaMemcpy2D(points, intrin.width * sizeof(float) * 3, dev_points, pitch, intrin.width * sizeof(float) * 3, intrin.height, cudaMemcpyDeviceToHost);
  //  printf("result: %d \n", result);
 /*   
    for (int i = 0; i < 3; i++) 
    {
        result = cudaStreamDestroy(streams[i]);
    }    
    */
    
  //  result = cudaStreamDestroy(stream1);
  //  result = cudaStreamDestroy(stream2);
  //  assert(result == cudaSuccess);
    
    cudaFree(dev_points);
    cudaFree(dev_depth);
    cudaFree(dev_intrin);
}

#endif
